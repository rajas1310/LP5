#include<iostream>
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>

#define BLOCK_SIZE 64

using namespace std;

void initialize_array(int *arr, int size){
	for(int i=0; i<size; i++){
		arr[i]=rand()%100;
	}
}

void add_cpu(int *arr1, int *arr2, int *res, int size){
	for(int i=0; i<size; i++){
		res[i]=arr1[i] + arr2[i]; 
	}
}

void print_matrix(int *arr, int size){
	for(int i=0; i<size; i++){
		cout<<arr[i]<<" ";
	}
}

__global__ void add_gpu(int *arr1, int *arr2, int *res, int size){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < size){
		res[id] = arr1[id] + arr2[id];
	}
}

int main(){
	int *arr1_cpu, *arr2_cpu, *res_cpu;
	int n;
	cout<<"Enter size of array: ";
	cin>>n;
	
	arr1_cpu = new int[n];
	arr2_cpu = new int[n];
	res_cpu = new int[n];
	
	initialize_array(arr1_cpu, n);
	initialize_array(arr2_cpu, n);
	
	/* CPU addition */
	auto start = chrono::high_resolution_clock::now();	
	add_cpu(arr1_cpu, arr2_cpu, res_cpu, n);
	auto end = chrono::high_resolution_clock::now();
	cout << "CPU Duration: "<< chrono::duration_cast<chrono::microseconds>(end-start).count()<< "microseconds"<<endl;
	
	/* GPU addition */
	
	int *arr1_gpu, *arr2_gpu, *res_gpu;
	dim3 dimGrid(n/BLOCK_SIZE);
	dim3 dimBlock(BLOCK_SIZE);
	
	hipMallocManaged(&arr1_gpu, n*sizeof(int));
	hipMallocManaged(&arr2_gpu, n*sizeof(int));
	hipMallocManaged(&res_gpu, n*sizeof(int));
	
	hipMemcpy(arr1_gpu, arr1_cpu, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(arr2_gpu, arr2_cpu, n*sizeof(int), hipMemcpyHostToDevice);
	start = chrono::high_resolution_clock::now();
	add_gpu<<<dimGrid,dimBlock>>>(arr1_gpu, arr2_gpu, res_gpu, n);
	end = chrono::high_resolution_clock::now();
	cout << "GPU Duration: "<< chrono::duration_cast<chrono::microseconds>(end-start).count()<< "microseconds"<<endl;
	
	hipMemcpy(arr1_cpu, arr1_gpu, n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(arr2_cpu, arr2_gpu, n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(res_cpu, res_gpu, n*sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(arr1_gpu);
	hipFree(arr2_gpu);
	hipFree(res_gpu);
	return 0;
}
