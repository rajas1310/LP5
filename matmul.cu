#include <iostream>
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

using namespace std;

void initialize_matrix(int *arr, int rows, int cols){
	for(int i=0; i<rows; i++){
		for(int j=0; j<cols; j++){
			arr[i*cols + j] = rand()%10;
		}
	}
}

void multiply_cpu(int *mat1, int *mat2, int *res, int rows, int common, int cols){
	for(int i=0; i<rows; i++){
		for(int j=0; j<cols; j++){
			int sum = 0;
			for(int k=0; k<common; k++){
				sum += mat1[i*common + k] * mat2[k*cols + j];
			}
			res[i*cols + j] = sum;
		}
	}
}


__global__ void multiply_gpu(int *mat1, int *mat2, int *res, int rows, int common, int cols){
	int r = blockIdx.x * blockDim.x + threadIdx.x;
	int c = blockIdx.y * blockDim.y + threadIdx.y;
	int sum = 0;
	
	if (r <rows && c <cols){
		for(int i=0; i<common; i++){
			sum += mat1[r*common + i] * mat2[i*cols + c]; 
		}
		res[r*cols + c] = sum;
	}
}

int main(){
	int R1, C1, R2, C2, R3, C3;
	cout<<"Enter no. of row for Mat1: ";
	cin>>R1;
	cout<<"Enter no. of columns for Mat1: ";
	cin>>C1;
	cout<<"Enter no. of columns for Mat2: ";
	cin>>C2;
	
	R2 = C1;
	R3 = R1;
	C3 = C2;
	
	int *mat1_cpu, *mat2_cpu, *res_cpu;
	mat1_cpu = new int[R1*C1];
	mat2_cpu = new int[R2*C2];
	res_cpu = new int[R3*C3];
	
	initialize_matrix(mat1_cpu, R1, C1);
	initialize_matrix(mat2_cpu, R2, C2);
	
	/* CPU multiplication */
	auto start = chrono::high_resolution_clock::now();
	multiply_cpu(mat1_cpu, mat2_cpu, res_cpu, R1, C1, C2);	
	auto end = chrono::high_resolution_clock::now();
	cout<<"CPU Duration: "<< chrono::duration_cast<chrono::microseconds>(end-start).count()<< " microseconds"<<endl;
	
	/* GPU multiplication */
	
	dim3 dimGrid(R3 / BLOCK_SIZE, C3 / BLOCK_SIZE);
	dim3 dimBlock(BLOCK_SIZE , BLOCK_SIZE);
	
	int *mat1_gpu, *mat2_gpu, *res_gpu;
	
	hipMallocManaged(&mat1_gpu, R1*C1*sizeof(int));
	hipMallocManaged(&mat2_gpu, R2*C2*sizeof(int));
	hipMallocManaged(&res_gpu, R3*C3*sizeof(int));
	
	hipMemcpy(mat1_gpu, mat1_cpu, R1*C1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mat2_gpu, mat2_cpu, R2*C2*sizeof(int), hipMemcpyHostToDevice);
	
	start = chrono::high_resolution_clock::now();
	multiply_gpu<<<dimGrid, dimBlock>>>(mat1_cpu, mat2_cpu, res_cpu, R1, C1, C2);
	end = chrono::high_resolution_clock::now();
	cout<<"GPU Duration: "<< chrono::duration_cast<chrono::microseconds>(end-start).count()<< " microseconds"<<endl;
	
	hipMemcpy(res_cpu, res_gpu, R3*C3*sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(mat1_gpu);
	hipFree(mat2_gpu);
	hipFree(res_gpu);
	
	return 0;
}
